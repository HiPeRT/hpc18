
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void mykernel()
{
  // Device function; runs on GPU!
  printf("Hello World! I am thread %d of block %d, and I am running on the GPGPU device\n", threadIdx.x, blockIdx.x);
}


int main()
{
  mykernel<<< 2, 4 >>>();

  hipDeviceSynchronize();

  printf("Hello World! I am running on host\n");
  return 0;
}
