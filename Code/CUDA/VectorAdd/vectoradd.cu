
#include <hip/hip_runtime.h>
#include <stdio.h>

// Compute vector sum C = A + B
void vecAdd(int *h_A, int *h_B, int *h_C, int n)
{
    int i;
    for (i = 0; i<n; i++)
       h_C[i] = h_A[i] + h_B[i];
}

int main()
{
    int n = 16;

    // Memory allocation for h_A, h_B, and h_C
    int *h_A = (int*) malloc(n * sizeof(int));
    int *h_B = (int*) malloc(n * sizeof(int));
    int *h_C = (int*) malloc(n * sizeof(int));
    
    // I/O to read h_A and h_B, N elements
    for(int i=0; i<n; i++)
    {
        h_A[i] = i;
        h_B[i] = 2;
    } 

    vecAdd(h_A, h_B, h_C, n);

    for(int i=0; i<n; i++)
    {
        printf("h_C[%d] is %d\n", i, h_C[i]);
    }
}

